#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include "colors.h"

#define MAX_ITERATIONS 65536
#define THREADS_PER_BLOCK 256

__device__ float autoPow(float x, float y)
{
	return powf(x, y);
}

__device__ double autoPow(double x, double y)
{
	return pow(x, y);
}

template<typename T>
__global__ void mandelbrotKernel(const int depthStart,
				const int depthEnd,
				const T zoomFactor,
				const int width,
				const int height,
				unsigned* const intensities)
{
	// Calculate the global thread index and exit if there is no work to do
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int pz = depthStart + idx / (height * width);
	const int py = (idx / width) % height;
	const int px = idx % width;

	if (idx >= (depthEnd - depthStart) * height * width) return;

	// Parameters for a really cool part that has great depth
	const T xCenter = static_cast<T>(-0.235125001);
	const T yCenter = static_cast<T>(0.827215);
	//const T xCenter = static_cast<T>(-0.598274455069517539539);
	//const T yCenter = static_cast<T>(0.663825928894102918143);
	const T baseVerticalRadius = static_cast<T>(1.0);//(0.00004);

	// This controls how much each depth zooms in, bounded by (0,1]. Lower values zoom faster.
	//const T zoomFactor = static_cast<T>(0.9);
	//const T zoomFactor = static_cast<T>(0.95);

	// Compute mandelbrot set
	const T yRadius = baseVerticalRadius * autoPow(zoomFactor, pz);
	const T xRadius = yRadius * width / height;

	const T y0 = yCenter + yRadius - static_cast<T>(2.0) * yRadius * py / (height - 1);
	const T x0 = xCenter - xRadius + static_cast<T>(2.0) * xRadius * px / (width - 1);

	T x = x0;
	T y = y0;
	T x2 = x * x;
	T y2 = y * y;
	unsigned iteration = 0;

	while (x2 + y2 <= static_cast<T>(4.0) && iteration < MAX_ITERATIONS)
	{
		y = static_cast<T>(2.0) * x * y + y0;
		x = x2 - y2 + x0;
		x2 = x * x;
		y2 = y * y;
		++iteration;
	}

	// Store intensity value for determining color later
	intensities[idx] = iteration;
}

template __global__ void mandelbrotKernel<float>(const int, const int, const float, const int, const int, unsigned* const);
template __global__ void mandelbrotKernel<double>(const int, const int, const double, const int, const int, unsigned* const);



void savePpmImage(const char* const filename, const int width, const int height, const unsigned* const intensities)
{
	// Open the file
	std::ofstream outfile(filename);

	// Write the header
	//outfile << "P3 " << width << " " << height << " " << MAX_ITERATIONS / 2 << "\n";
	outfile << "P3 " << width << " " << height << " " << 255 << "\n";

	// Write pixel information
	for (int y = 0; y < height; ++y)
	{
		const int yidx = y * width;

		for (int x = 0; x < width; ++x)
		{
			//const std::array<unsigned, 3> color = mapIntensityToColor(intensities[yidx + x]);
			//outfile << color[0] << " " << color[1] << " " << color[2] << " ";
			const unsigned i = intensities[yidx + x] % 256;
			outfile << djl70::paletteRed[i] << " " << djl70::paletteGreen[i] << " " << djl70::paletteBlue[i] << " ";
		}

		outfile << "\n";
	}

	outfile.close();
}



void printUsage(const char* const programName)
{
	std::cerr << "Usage: " << programName << " <depths> <width> [save_output [depth_start [zoom_factor]]]\n"
		<< "Description: Computes the Mandelbrot set using CUDA\n"
		<< "Argument combinations worth trying:\n"
		<< "    " << programName << " 256 120 1\n"
		<< "    " << programName << " 16 600 1\n"
		<< "    " << programName << " 64 2400 0\n"
		<< "    " << programName << " 64 2400 0 64\n"
		<< "    " << programName << " 512 600 0 0 0.99\n"
		<< "\n"
		<< "depths:      The number of 'layers' to process (each layer 'zooms in' to the set)\n"
		<< "width:       The width of the images to process (height is set automatically)\n"
		<< "save_output: (optional, default 0) Set to 1 to save the processed images (note: not recommended beyond depths=64 and width=600, because otherwise saving the images may require a lot of time and disk space)\n"
		<< "depth_start: (optional, default 0) The zero-based 'layer' to begin processing at, inclusive\n"
		<< "zoom_factor: (optional, default 0.9) A value (0 < zoom_factor <= 1) deciding how much to zoom for each 'layer'. Higher values produce slower zooms"
		<< std::endl;
}

int main(int argc, char* argv[])
{
	// Verify command line arguments
	if (argc < 3)
	{
		printUsage(argv[0]);
		return -1;
	}
	const int depths = atoi(argv[1]);
	if (depths < 1)
	{
		std::cerr << "Error: arg 'depths' must be at least 1" << std::endl;
		return -1;
	}
	const int width = atoi(argv[2]);
	if (width < 2)
	{
		std::cerr << "Error: arg 'width' must be at least 2" << std::endl;
		return -1;
	}
	const int height = (float)width * 2.0f / 3.0f;
	const bool doSaveImages = (argc >= 4) && (atoi(argv[3]) == 1);
	const int depthStart = (argc >= 5) ? atoi(argv[4]) : 0;
	if (depthStart < 0)
	{
		std::cerr << "Error: arg 'depth_start' must be at least 0" << std::endl;
		return -1;
	}
	const int depthEnd = depthStart + depths;
	const double zoomFactor = (argc >= 6) ? atof(argv[5]) : 0.9;
	std::cout << "'depths' = " << depths
		<< "\n'width' = " << width
		<< "\n'height' = " << height
		<< "\n'save_output' = " << doSaveImages
		<< "\n'depth_start' = " << depthStart << " (inclusive)"
		<< "\n'depth_end' = " << depthEnd << " (exclusive)"
		<< "\n'zoom_factor' = " << zoomFactor
		<< std::endl;

	// Allocate host memory
	const int n = depths * height * width;
	unsigned* h_intensities = new unsigned[n];

	// Allocate device memory
	unsigned* d_intensities;
	if (hipSuccess != hipMalloc((void**)&d_intensities, sizeof(unsigned) * n))
	{
		delete[] h_intensities;
		std::cerr << "Error: failed to allocate device memory" << std::endl;
		return -1;
	}

	// Launch GPU kernel and begin timing
	timeval start, end;
	if (doSaveImages || depthEnd > 128)
	{
		std::cout << "'depth_end' > 128 or 'save_output' = 1, launching kernel with double precision to ensure high quality output" << std::endl;
		gettimeofday(&start, NULL);
		mandelbrotKernel<double><<<(n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(depthStart, depthEnd, zoomFactor, width, height, d_intensities);
	}
	else
	{
		std::cout << "'depth_end' <= 128 and 'save_output' = 0, launching kernel with single precision to ensure the best performance" << std::endl;
		gettimeofday(&start, NULL);
		mandelbrotKernel<float><<<(n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(depthStart, depthEnd, zoomFactor, width, height, d_intensities);
	}
	hipDeviceSynchronize();

	// End timing
	gettimeofday(&end, NULL);
	const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
	std::cout << "Kernel runtime: " << std::fixed << std::setprecision(4) << runtime << " s" << std::endl;

	// Check for errors from the kernel
	hipError_t e = hipGetLastError();
	if (hipSuccess != e)
	{
		delete[] h_intensities;
		hipFree(d_intensities);
		std::cerr << "CUDA error " << e << ": " << hipGetErrorString(e) << std::endl;
		return -1;
	}

	// Save images if desired
	if (doSaveImages)
	{
		// Copy results to the host
		if (hipSuccess != hipMemcpy(h_intensities, d_intensities, sizeof(unsigned) * n, hipMemcpyDeviceToHost))
		{
			delete[] h_intensities;
			hipFree(d_intensities);
			std::cerr << "Error: failed to copy from device to host" << std::endl;
			return -1;
		}

		std::cout << "Saving output images, please wait..." << std::endl;
		for (int i = depthStart; i < depthEnd; ++i)
		{
			std::stringstream filename;
			filename << "mandelbrot" << std::setw(3) << std::setfill('0') << i << std::setfill(' ') << ".ppm";
			savePpmImage(filename.str().c_str(), width, height, &h_intensities[(i - depthStart) * height * width]);
		}
		std::cout << "Output images successfully saved as .ppm files\n"
			<< "Try running 'convert -delay 10 mandelbrot*ppm mandelbrot.gif' to create an animation"
			<< std::endl;
	}

	// Free memory
	delete[] h_intensities;
	hipFree(d_intensities);

	return 0;
}
